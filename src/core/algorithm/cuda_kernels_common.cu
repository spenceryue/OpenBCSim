#include "hip/hip_runtime.h"
/*
Copyright (c) 2015, Sigurd Storve
All rights reserved.

Licensed under the BSD license.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "cuda_helpers.h"
#include "cuda_kernels_common.cuh"
#include "" // for removing annoying MSVC intellisense error messages
#include <hip/hip_complex.h>                // for hipCmulf()
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math_functions.h> // for sincosf()

template <bool normalize>
__global__ void MultiplyFftKernel (hipfftComplex *time_proj_fft, const hipfftComplex *filter_fft, int num_samples)
{
  const auto global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_idx < num_samples)
  {
    hipfftComplex a = time_proj_fft[global_idx];
    hipfftComplex b = filter_fft[global_idx];
    if (normalize)
    {
      time_proj_fft[global_idx] = make_float2 (a.x * b.x / num_samples - a.y * b.y / num_samples,
                                               a.x * b.y / num_samples + a.y * b.x / num_samples);
    }
    else
    {
      time_proj_fft[global_idx] = make_float2 (a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
    }
  }
}

// Export explicit instantiations for linking
template __global__ void MultiplyFftKernel<false> (hipfftComplex *time_proj_fft, const hipfftComplex *filter_fft, int num_samples);
template __global__ void MultiplyFftKernel<true> (hipfftComplex *time_proj_fft, const hipfftComplex *filter_fft, int num_samples);

__global__ void ScaleSignalKernel (hipfftComplex *signal, float factor, int num_samples)
{
  const auto global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_idx < num_samples)
  {
    hipfftComplex c = signal[global_idx];
    signal[global_idx] = make_float2 (c.x * factor, c.y * factor);
  }
}

__global__ void DemodulateKernel (hipComplex *signal, float w, int stop_index, int radial_decimation)
{
  const auto global_idx = (blockIdx.x * blockDim.x + threadIdx.x) * radial_decimation;
  if (global_idx < stop_index)
  {
    // exp(-i*w*n) = cos(w*n) - i*sin(w*n)
    float sin_value, cos_value;
    sincosf (w * global_idx, &sin_value, &cos_value);
    const auto c = make_hipComplex (cos_value, -sin_value);

    signal[global_idx] = hipCmulf (signal[global_idx], c);
  }
}

__global__ void AddNoiseKernel (hipComplex *signal, hipComplex *noise, int num_samples)
{
  const auto global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_idx < num_samples)
  {
    signal[global_idx] = make_hipComplex (signal[global_idx].x + noise[global_idx].x, signal[global_idx].y + noise[global_idx].y);
  }
}
